#include "hip/hip_runtime.h"
/*----< find_nearest_cluster() >---------------------------------------------*/
__global__ static
void find_nearest_cluster(int numCoords,
                          int numObjs,
                          int numClusters,
                          double *deviceobjects,           //  [numCoords][numObjs]
                          int *devicenewClusterSize,           //  [numClusters]
                          double *devicenewClusters,    //  [numCoords][numClusters]
                          double *deviceClusters,    //  [numCoords][numClusters]
                          int *deviceMembership,          //  [numObjs]
                          double *devdelta) {
  extern __shared__ double shmem_total[];
  double *shmemClusters = shmem_total;
  double *delta_reduce_buff = shmem_total + numClusters * numCoords;
  /* TODO: copy me from shared version... */
  int no_cluster, i;
  
  //use local_id because shared memory is per thread block
  for (no_cluster = threadIdx.x; no_cluster < numClusters; no_cluster+=blockDim.x) {
    for (i = 0; i < numCoords; i++) {
      shmemClusters[i * numClusters + no_cluster] = deviceClusters[i * numClusters + no_cluster];
    }
  }
  __syncthreads();

  /* Get the global ID of the thread. */
  int tid = get_tid();

  /* TODO: copy me from shared version... */
  if (tid < numObjs) {

    /* TODO: copy me from shared version... */
    int index;
    double dist, min_dist;

    /* find the cluster id that has min distance to object */
    index = 0;
    /* TODO: call min_dist = euclid_dist_2(...) with correct objectId/clusterId using clusters in shmem*/
    min_dist = euclid_dist_2_transpose(numCoords, numObjs, numClusters, deviceobjects, shmemClusters, tid, 0);
    for (i = 1; i < numClusters; i++) {
      /* TODO: call dist = euclid_dist_2(...) with correct objectId/clusterId using clusters in shmem*/
      dist = euclid_dist_2_transpose(numCoords, numObjs, numClusters, deviceobjects, shmemClusters, tid, i);

      /* no need square root */
      if (dist < min_dist) { /* find the min and its array index */
        min_dist = dist;
        index = i;
      }
    }

    if (deviceMembership[tid] != index) {
      delta_reduce_buff[threadIdx.x] = 1.0;
    }
    else {
      delta_reduce_buff[threadIdx.x] = 0.0;
    }

    /* assign the deviceMembership to object objectId */
    deviceMembership[tid] = index;

    /* TODO: Replacing (*devdelta)+= 1.0; with reduction:
      - each thread updates the single element of delta_reduce_buff
      corresponding to its local id (threadIdx.x) -> 1.0 if membership changes, otherwise 0.
      - Then, ensuring delta_reduce_buff is fully updated, its containts must be summed in delta_reduce_buff[0]
      either by one thread (lower perf) or with a tree-based reduction (similar to dot reduction example in slides)
      - Finally, delta_reduce_buff[0] (local value in block) must be added to devdelta (global delta value), ensuring write dependencies!
    */

    /* TODO: additional steps for calculating new centroids in GPU? */
    atomicAdd(&devicenewClusterSize[index], 1);
    for (i = 0; i < numCoords; i++) {
      atomicAdd(&devicenewClusters[i * numClusters + index], deviceobjects[i * numObjs + tid]);
    }

    __syncthreads();
    //after everyone in the block is finished do the tree update of delta
    i = blockDim.x / 2;
    while (i != 0) {
      if (threadIdx.x < i) delta_reduce_buff[threadIdx.x] += delta_reduce_buff[threadIdx.x + i];
      __syncthreads();
      i /= 2;
    }
    if (threadIdx.x == 0) atomicAdd(devdelta, delta_reduce_buff[0]);
  }
}

