  do {
    timing_internal = wtime();

    /* GPU part: calculate new memberships */

    timing_transfers = wtime();
    /* TODO: Copy clusters to deviceClusters
    checkCuda(hipMemcpy(...)); */
    checkCuda(hipMemcpy(deviceClusters, dimClusters[0], numClusters * numCoords * sizeof(double), hipMemcpyHostToDevice));
    transfers_time += wtime() - timing_transfers;

    checkCuda(hipMemset(dev_delta_ptr, 0, sizeof(double)));

    //printf("Launching find_nearest_cluster Kernel with grid_size = %d, block_size = %d, shared_mem = %d KB\n", numClusterBlocks, numThreadsPerClusterBlock, clusterBlockSharedDataSize/1000);
    timing_gpu = wtime();
    find_nearest_cluster
    <<< numClusterBlocks, numThreadsPerClusterBlock, clusterBlockSharedDataSize >>>
            (numCoords, numObjs, numClusters,
             deviceObjects, deviceClusters, deviceMembership, dev_delta_ptr);

    hipDeviceSynchronize();
    checkLastCudaError();
    gpu_time += wtime() - timing_gpu;
    //printf("Kernels complete for itter %d, updating data in CPU\n", loop);

    timing_transfers = wtime();
    /* TODO: Copy deviceMembership to membership
        checkCuda(hipMemcpy(...)); */
        checkCuda(hipMemcpy(membership, deviceMembership, numObjs * sizeof(int), hipMemcpyDeviceToHost));

    /* TODO: Copy dev_delta_ptr to &delta
      checkCuda(hipMemcpy(...)); */
      checkCuda(hipMemcpy(&delta, dev_delta_ptr, sizeof(double), hipMemcpyDeviceToHost));
    transfers_time += wtime() - timing_transfers;

    /* CPU part: Update cluster centers*/

    timing_cpu = wtime();
    for (i = 0; i < numObjs; i++) {
      /* find the array index of nestest cluster center */
      index = membership[i];

      /* update new cluster centers : sum of objects located within */
      newClusterSize[index]++;
      for (j = 0; j < numCoords; j++)
        newClusters[j][index] += objects[i * numCoords + j];
    }

    /* average the sum and replace old cluster centers with newClusters */
    for (i = 0; i < numClusters; i++) {
      for (j = 0; j < numCoords; j++) {
        if (newClusterSize[i] > 0)
          dimClusters[j][i] = newClusters[j][i] / newClusterSize[i];
        newClusters[j][i] = 0.0;   /* set back to 0 */
      }
      newClusterSize[i] = 0;   /* set back to 0 */
    }

    delta /= numObjs;
    //printf("delta is %f - ", delta);
    loop++;
    //printf("completed loop %d\n", loop);
    cpu_time += wtime() - timing_cpu;

    timing_internal = wtime() - timing_internal;
    if (timing_internal < timer_min) timer_min = timing_internal;
    if (timing_internal > timer_max) timer_max = timing_internal;
  } while (delta > threshold && loop < loop_threshold);

  /*TODO: Update clusters using dimClusters. Be carefull of layout!!! clusters[numClusters][numCoords] vs dimClusters[numCoords][numClusters] */
  for (i = 0; i < numClusters; i++) {
    for (j = 0; j < numCoords; j++) {
      clusters[i*numCoords + j] = dimClusters[j][i];
    }
  }
