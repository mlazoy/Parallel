  do {
    timing_internal = wtime();
    checkCuda(hipMemset(dev_delta_ptr, 0, sizeof(double)));
    timing_gpu = wtime();
    //printf("Launching find_nearest_cluster Kernel with grid_size = %d, block_size = %d, shared_mem = %d KB\n", numClusterBlocks, numThreadsPerClusterBlock, clusterBlockSharedDataSize/1000);
    /* TODO: change invocation if extra parameters needed
    find_nearest_cluster
        <<< numClusterBlocks, numThreadsPerClusterBlock, clusterBlockSharedDataSize >>>
        (numCoords, numObjs, numClusters,
         deviceObjects, devicenewClusterSize, devicenewClusters, deviceClusters, deviceMembership, dev_delta_ptr);
    */
    find_nearest_cluster<<< numClusterBlocks, numThreadsPerClusterBlock, clusterBlockSharedDataSize >>>
        (numCoords, numObjs, numClusters, 
          deviceObjects, devicenewClusterSize, devicenewClusters, deviceClusters, deviceMembership, dev_delta_ptr);

    hipDeviceSynchronize();
    checkLastCudaError();

    gpu_time += wtime() - timing_gpu;

    //printf("Kernels complete for itter %d, updating data in CPU\n", loop);

    timing_transfers = wtime();
    /* TODO: Copy dev_delta_ptr to &delta
      checkCuda(hipMemcpy(...)); */
      checkCuda(hipMemcpy(&delta, dev_delta_ptr, sizeof(double), hipMemcpyDeviceToHost));
    transfers_time += wtime() - timing_transfers;

    const unsigned int update_centroids_block_sz = (numCoords * numClusters > blockSize) ? blockSize : numCoords *
                                                                                                       numClusters;  /* TODO: can use different blocksize here if deemed better */
    const unsigned int update_centroids_dim_sz = (numCoords * numClusters + update_centroids_block_sz - 1) / update_centroids_block_sz; /* TODO: calculate dim for "update_centroids" */
    timing_gpu = wtime();
    /* TODO: use dim for "update_centroids" and fire it
     	update_centroids<<< update_centroids_dim_sz, update_centroids_block_sz, 0 >>>
            (numCoords, numClusters, devicenewClusterSize, devicenewClusters, deviceClusters);  */
      update_centroids<<< update_centroids_dim_sz, update_centroids_block_sz, 0 >>>
      (numCoords, numClusters, devicenewClusterSize, devicenewClusters, deviceClusters); 
    hipDeviceSynchronize();
    checkLastCudaError();
    gpu_time += wtime() - timing_gpu;

    timing_cpu = wtime();
    delta /= numObjs;
    //printf("delta is %f - ", delta);
    loop++;
    //printf("completed loop %d\n", loop);
    cpu_time += wtime() - timing_cpu;

    timing_internal = wtime() - timing_internal;
    if (timing_internal < timer_min) timer_min = timing_internal;
    if (timing_internal > timer_max) timer_max = timing_internal;
  } while (delta > threshold && loop < loop_threshold);

