#include "hip/hip_runtime.h"
__global__ static
void find_nearest_cluster(int numCoords,
                          int numObjs,
                          int numClusters,
                          double *objects,           //  [numCoords][numObjs]
                          double *deviceClusters,    //  [numCoords][numClusters]
                          int *deviceMembership,          //  [numObjs]
                          double *devdelta) {
  extern __shared__ double shmemClusters[];

  /* TODO: Copy deviceClusters to shmemClusters so they can be accessed faster.
    BEWARE: Make sure operations is complete before any thread continues... */
  int no_cluster, i;
  
  //use local_id because shared memory is per thread block
  for (no_cluster = threadIdx.x; no_cluster < numClusters; no_cluster+=blockDim.x) {
    for (i = 0; i < numCoords; i++) {
      shmemClusters[i * numClusters + no_cluster] = deviceClusters[i * numClusters + no_cluster];
    }
  }
  __syncthreads();

  /* Get the global ID of the thread. */
  int tid = get_tid();

  /* TODO: Maybe something is missing here... should all threads run this? */
  if (tid < numObjs) {
    int index;
    double dist, min_dist;

    /* find the cluster id that has min distance to object */
    index = 0;
    /* TODO: call min_dist = euclid_dist_2(...) with correct objectId/clusterId using clusters in shmem*/
    min_dist = euclid_dist_2_transpose(numCoords, numObjs, numClusters, objects, shmemClusters, tid, 0);
    for (i = 1; i < numClusters; i++) {
      /* TODO: call dist = euclid_dist_2(...) with correct objectId/clusterId using clusters in shmem*/
      dist = euclid_dist_2_transpose(numCoords, numObjs, numClusters, objects, shmemClusters, tid, i);

      /* no need square root */
      if (dist < min_dist) { /* find the min and its array index */
        min_dist = dist;
        index = i;
      }
    }

    if (deviceMembership[tid] != index) {
      /* TODO: Maybe something is missing here... is this write safe? */
      atomicAdd(devdelta, 1.0);
    }

    /* assign the deviceMembership to object objectId */
    deviceMembership[tid] = index;
  }

}

