
#include <hip/hip_runtime.h>
__host__ __device__ inline static
double euclid_dist_2_transpose(int numCoords,
                               int numObjs,
                               int numClusters,
                               double *objects,     // [numCoords][numObjs]
                               double *clusters,    // [numCoords][numClusters]
                               int objectId,
                               int clusterId) {
  int i;
  double ans = 0.0, diff;

  /* TODO: Calculate the euclid_dist of elem=objectId of objects from elem=clusterId from clusters, but for column-base format!!! */
  for(i = 0; i < numCoords; i++) {
    diff = objects[i*numObjs+ objectId] - clusters[i*numClusters + clusterId];
    ans += diff * diff;
  }

  return (ans);
}
