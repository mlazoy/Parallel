#include "hip/hip_runtime.h"
__global__ static
void find_nearest_cluster(int numCoords,
                          int numObjs,
                          int numClusters,
                          double *deviceobjects,           //  [numCoords][numObjs]
/*                          
                          TODO: If you choose to do (some of) the new centroid calculation here, you will need some extra parameters here (from "update_centroids").
*/
                          int *devicenewClusterSize,           //  [numClusters]
                          double *devicenewClusters,    //  [numCoords][numClusters]
                          double *deviceClusters,    //  [numCoords][numClusters]
                          int *deviceMembership,          //  [numObjs]
                          double *devdelta) {
  extern __shared__ double shmemClusters[];

  /* TODO: copy me from shared version... */
  int no_cluster, i;
  
  //use local_id because shared memory is per thread block
  for (no_cluster = threadIdx.x; no_cluster < numClusters; no_cluster+=blockDim.x) {
    for (i = 0; i < numCoords; i++) {
      shmemClusters[i * numClusters + no_cluster] = deviceClusters[i * numClusters + no_cluster];
    }
  }
  __syncthreads();

  /* Get the global ID of the thread. */
  int tid = get_tid();

  /* TODO: copy me from shared version... */
  if (tid < numObjs) {
    int index;
    double dist, min_dist;

    /* find the cluster id that has min distance to object */
    index = 0;
    /* TODO: call min_dist = euclid_dist_2(...) with correct objectId/clusterId using clusters in shmem*/
    min_dist = euclid_dist_2_transpose(numCoords, numObjs, numClusters, deviceobjects, shmemClusters, tid, 0);
    for (i = 1; i < numClusters; i++) {
      /* TODO: call dist = euclid_dist_2(...) with correct objectId/clusterId using clusters in shmem*/
      dist = euclid_dist_2_transpose(numCoords, numObjs, numClusters, deviceobjects, shmemClusters, tid, i);

      /* no need square root */
      if (dist < min_dist) { /* find the min and its array index */
        min_dist = dist;
        index = i;
      }
    }

    if (deviceMembership[tid] != index) {
      /* TODO: Maybe something is missing here... is this write safe? */
      atomicAdd(devdelta, 1.0);
    }

    /* assign the deviceMembership to object objectId */
    deviceMembership[tid] = index;

    /* TODO: additional steps for calculating new centroids in GPU? */
    //we chose to update the size and do the add here
    //the division and the actual new Coords will be in update controids
    atomicAdd(&devicenewClusterSize[index], 1);
    for (i = 0; i < numCoords; i++) 
      atomicAdd(&devicenewClusters[i * numClusters + index], deviceobjects[i * numObjs + tid]);
  }
}

__global__ static
void update_centroids(int numCoords,
                      int numClusters,
                      int *devicenewClusterSize,           //  [numClusters]
                      double *devicenewClusters,    //  [numCoords][numClusters]
                      double *deviceClusters)    //  [numCoords][numClusters])
{

  /* TODO: additional steps for calculating new centroids in GPU? */
  int tid = get_tid();

  if (tid < numCoords * numClusters) {
    /*run through all the elements, just divide by the size of the clusters
    indexing of the 1d colummn based devicenewClusters is i*numClusters + j
    so the index of the current cluster is the j, and i the Coords 
    so the index of the current clusters is (i*numClusters + j) % numClusters
    here the tid runs all the array increasingly so it is i*numClusters + j
    */
    deviceClusters[tid] = devicenewClusters[tid] / devicenewClusterSize[tid % numClusters];
    //reset devicenewClusters after updating deviceClusters
    devicenewClusters[tid] = 0.0;
  }
  __syncthreads();
  //reset devicenewClusterSize as well
  if (tid < numClusters) {
    devicenewClusterSize[tid] = 0;
  }

}

