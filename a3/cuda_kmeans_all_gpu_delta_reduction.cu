#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#include "kmeans.h"
#include "alloc.h"
#include "error.h"

#ifdef __HIPCC__
inline void checkCuda(hipError_t e) {
    if (e != hipSuccess) {
        // hipGetErrorString() isn't always very helpful. Look up the error
        // number in the hipError_t enum in hip/driver_types.h in the CUDA includes
        // directory for a better explanation.
        error("CUDA Error %d: %s\n", e, hipGetErrorString(e));
    }
}

inline void checkLastCudaError() {
    checkCuda(hipGetLastError());
}
#endif

__device__ int get_tid() {
  /* TODO: Copy from full-offload */
  return blockDim.x*blockIdx.x + threadIdx.x;
}

/* square of Euclid distance between two multi-dimensional points using column-base format */
__host__ __device__ inline static
double euclid_dist_2_transpose(int numCoords,
                               int numObjs,
                               int numClusters,
                               double *objects,     // [numCoords][numObjs]
                               double *clusters,    // [numCoords][numClusters]
                               int objectId,
                               int clusterId) {
  int i;
  double ans = 0.0, diff;

  /* TODO: Copy from full-offload */
  for(i = 0; i < numCoords; i++) {
    diff = objects[i*numObjs+ objectId] - clusters[i*numClusters + clusterId];
    ans += diff * diff;
  }

  return (ans);
}

/*----< find_nearest_cluster() >---------------------------------------------*/
__global__ static
void find_nearest_cluster(int numCoords,
                          int numObjs,
                          int numClusters,
                          double *deviceobjects,           //  [numCoords][numObjs]
                          int *devicenewClusterSize,           //  [numClusters]
                          double *devicenewClusters,    //  [numCoords][numClusters]
                          double *deviceClusters,    //  [numCoords][numClusters]
                          int *deviceMembership,          //  [numObjs]
                          double *devdelta) {
  extern __shared__ double shmem_total[];
  double *shmemClusters = shmem_total;
  double *delta_reduce_buff = shmem_total + numClusters * numCoords;
  /* TODO: copy me from shared version... */
  int no_cluster, i;
  
  //use local_id because shared memory is per thread block
  for (no_cluster = threadIdx.x; no_cluster < numClusters; no_cluster+=blockDim.x) {
    for (i = 0; i < numCoords; i++) {
      shmemClusters[i * numClusters + no_cluster] = deviceClusters[i * numClusters + no_cluster];
    }
  }
  __syncthreads();

  /* Get the global ID of the thread. */
  int tid = get_tid();

  /* TODO: copy me from shared version... */
  if (tid < numObjs) {

    /* TODO: copy me from shared version... */
    int index;
    double dist, min_dist;

    /* find the cluster id that has min distance to object */
    index = 0;
    /* TODO: call min_dist = euclid_dist_2(...) with correct objectId/clusterId using clusters in shmem*/
    min_dist = euclid_dist_2_transpose(numCoords, numObjs, numClusters, deviceobjects, shmemClusters, tid, 0);
    for (i = 1; i < numClusters; i++) {
      /* TODO: call dist = euclid_dist_2(...) with correct objectId/clusterId using clusters in shmem*/
      dist = euclid_dist_2_transpose(numCoords, numObjs, numClusters, deviceobjects, shmemClusters, tid, i);

      /* no need square root */
      if (dist < min_dist) { /* find the min and its array index */
        min_dist = dist;
        index = i;
      }
    }

    if (deviceMembership[tid] != index) {
      delta_reduce_buff[threadIdx.x] = 1.0;
    }
    else {
      delta_reduce_buff[threadIdx.x] = 0.0;
    }

    /* assign the deviceMembership to object objectId */
    deviceMembership[tid] = index;

    /* TODO: Replacing (*devdelta)+= 1.0; with reduction:
      - each thread updates the single element of delta_reduce_buff
      corresponding to its local id (threadIdx.x) -> 1.0 if membership changes, otherwise 0.
      - Then, ensuring delta_reduce_buff is fully updated, its containts must be summed in delta_reduce_buff[0]
      either by one thread (lower perf) or with a tree-based reduction (similar to dot reduction example in slides)
      - Finally, delta_reduce_buff[0] (local value in block) must be added to devdelta (global delta value), ensuring write dependencies!
    */

    /* TODO: additional steps for calculating new centroids in GPU? */
    atomicAdd(&devicenewClusterSize[index], 1);
    for (i = 0; i < numCoords; i++) {
      atomicAdd(&devicenewClusters[i * numClusters + index], deviceobjects[i * numObjs + tid]);
    }

    __syncthreads();
    //after everyone in the block is finished do the tree update of delta
    i = blockDim.x / 2;
    while (i != 0) {
      if (threadIdx.x < i) delta_reduce_buff[threadIdx.x] += delta_reduce_buff[threadIdx.x + i];
      __syncthreads();
      i /= 2;
    }
    if (threadIdx.x == 0) atomicAdd(devdelta, delta_reduce_buff[0]);
  }
}

__global__ static
void update_centroids(int numCoords,
                      int numClusters,
                      int *devicenewClusterSize,           //  [numClusters]
                      double *devicenewClusters,    //  [numCoords][numClusters]
                      double *deviceClusters)    //  [numCoords][numClusters])
{
  /* TODO: Copy from full-offload */
  int tid = get_tid();

  if (tid < numCoords * numClusters) {
    /*run through all the elements, just divide by the size of the clusters
    indexing of the 1d colummn based devicenewClusters is i*numClusters + j
    so the index of the current cluster is the j, and i the Coords 
    so the index of the current clusters is (i*numClusters + j) % numClusters
    here the tid runs all the array increasingly so it is i*numClusters + j
    */
    deviceClusters[tid] = devicenewClusters[tid] / devicenewClusterSize[tid % numClusters];
    //reset devicenewClusters after updating deviceClusters
    devicenewClusters[tid] = 0.0;
  }
  __syncthreads();
  //reset devicenewClusterSize as well
  if (tid < numClusters) {
    devicenewClusterSize[tid] = 0;
  }
}

//
//  ----------------------------------------
//  DATA LAYOUT
//
//  objects         [numObjs][numCoords]
//  clusters        [numClusters][numCoords]
//  dimObjects      [numCoords][numObjs]
//  dimClusters     [numCoords][numClusters]
//  newClusters     [numCoords][numClusters]
//  deviceObjects   [numCoords][numObjs]
//  deviceClusters  [numCoords][numClusters]
//  ----------------------------------------
//
/* return an array of cluster centers of size [numClusters][numCoords]       */
void kmeans_gpu(double *objects,      /* in: [numObjs][numCoords] */
                int numCoords,    /* no. features */
                int numObjs,      /* no. objects */
                int numClusters,  /* no. clusters */
                double threshold,    /* % objects change membership */
                long loop_threshold,   /* maximum number of iterations */
                int *membership,   /* out: [numObjs] */
                double *clusters,   /* out: [numClusters][numCoords] */
                int blockSize) {
  double timing = wtime(), timing_internal, timer_min = 1e42, timer_max = 0;
  double timing_gpu, timing_cpu, timing_transfers, transfers_time = 0.0, cpu_time = 0.0, gpu_time = 0.0;
  int loop_iterations = 0;
  int i, j, index, loop = 0;
  double delta = 0, *dev_delta_ptr;          /* % of objects change their clusters */
  /* TODO: Copy me from transpose version*/
  double **dimObjects = (double**) calloc_2d(numCoords, numObjs, sizeof(double)); //calloc_2d(...) -> [numCoords][numObjs]
  double **dimClusters = (double**) calloc_2d(numCoords, numClusters, sizeof(double));  //calloc_2d(...) -> [numCoords][numClusters]
  double **newClusters = (double**) calloc_2d(numCoords, numClusters, sizeof(double));  //calloc_2d(...) -> [numCoords][numClusters]

  printf("\n|-----------Full-offload Delta Reduction GPU Kmeans------------|\n\n");

  /* TODO: Copy me from transpose version*/
  for (i = 0; i < numObjs; i++) {
    for (j = 0; j < numCoords; j++) {
      dimObjects[j][i] = objects[i*numCoords+ j];
    }
  }

  double *deviceObjects;
  double *deviceClusters, *devicenewClusters;
  int *deviceMembership;
  int *devicenewClusterSize; /* [numClusters]: no. objects assigned in each new cluster */

  /* pick first numClusters elements of objects[] as initial cluster centers*/
  for (i = 0; i < numCoords; i++) {
    for (j = 0; j < numClusters; j++) {
      dimClusters[i][j] = dimObjects[i][j];
    }
  }

  /* initialize membership[] */
  for (i = 0; i < numObjs; i++) membership[i] = -1;

  timing = wtime() - timing;
  printf("t_alloc: %lf ms\n\n", 1000 * timing);
  timing = wtime();
  const unsigned int numThreadsPerClusterBlock = (numObjs > blockSize) ? blockSize : numObjs;
  const unsigned int numClusterBlocks = (numObjs + numThreadsPerClusterBlock - 1) / numThreadsPerClusterBlock; /* TODO: Calculate Grid size, e.g. number of blocks. */

  /*	Define the shared memory needed per block.
    - BEWARE: Also add extra shmem for delta buffer.
      - BEWARE: We can overrun our shared memory here if there are too many
      clusters or too many coordinates!
      - This can lead to occupancy problems or even inability to run.
      - Your exercise implementation is not requested to account for that (e.g. always assume deviceClusters fit in shmemClusters */
  const unsigned int clusterBlockSharedDataSize = numClusters * numCoords * sizeof(double) + numThreadsPerClusterBlock * sizeof(double);

  hipDeviceProp_t deviceProp;
  int deviceNum;
  hipGetDevice(&deviceNum);
  hipGetDeviceProperties(&deviceProp, deviceNum);

  if (clusterBlockSharedDataSize > deviceProp.sharedMemPerBlock) {
    error("Your CUDA hardware has insufficient block shared memory to hold all cluster centroids\n");
  }

  checkCuda(hipMalloc(&deviceObjects, numObjs * numCoords * sizeof(double)));
  checkCuda(hipMalloc(&deviceClusters, numClusters * numCoords * sizeof(double)));
  checkCuda(hipMalloc(&devicenewClusters, numClusters * numCoords * sizeof(double)));
  checkCuda(hipMalloc(&devicenewClusterSize, numClusters * sizeof(int)));
  checkCuda(hipMalloc(&deviceMembership, numObjs * sizeof(int)));
  checkCuda(hipMalloc(&dev_delta_ptr, sizeof(double)));

  timing = wtime() - timing;
  printf("t_alloc_gpu: %lf ms\n\n", 1000 * timing);
  timing = wtime();

  checkCuda(hipMemcpy(deviceObjects, dimObjects[0],
                       numObjs * numCoords * sizeof(double), hipMemcpyHostToDevice));
  checkCuda(hipMemcpy(deviceMembership, membership,
                       numObjs * sizeof(int), hipMemcpyHostToDevice));
  checkCuda(hipMemcpy(deviceClusters, dimClusters[0],
                       numClusters * numCoords * sizeof(double), hipMemcpyHostToDevice));
  checkCuda(hipMemset(devicenewClusterSize, 0, numClusters * sizeof(int)));
  //because we have do while we need to make sure the first time newClusters are set to 0
  //that is needed because the logic we followed is reseting newClusters, newClusterSize in update_centroids
  checkCuda(hipMemset(devicenewClusters, 0.0, numClusters * numCoords * sizeof(double)));
  free(dimObjects[0]);
  timing = wtime() - timing;
  printf("t_get_gpu: %lf ms\n\n", 1000 * timing);
  timing = wtime();

  do {
    timing_internal = wtime();
    checkCuda(hipMemset(dev_delta_ptr, 0, sizeof(double)));
    timing_gpu = wtime();

    //printf("Launching find_nearest_cluster Kernel with grid_size = %d, block_size = %d, shared_mem = %d KB\n", numClusterBlocks, numThreadsPerClusterBlock, clusterBlockSharedDataSize/1000);
    /* TODO: change invocation if extra parameters needed
    find_nearest_cluster
      <<< numClusterBlocks, numThreadsPerClusterBlock, clusterBlockSharedDataSize >>>
      (numCoords, numObjs, numClusters,
       deviceObjects, devicenewClusterSize, devicenewClusters, deviceClusters, deviceMembership, dev_delta_ptr);
    */
    find_nearest_cluster<<< numClusterBlocks, numThreadsPerClusterBlock, clusterBlockSharedDataSize >>>
        (numCoords, numObjs, numClusters, 
          deviceObjects, devicenewClusterSize, devicenewClusters, deviceClusters, deviceMembership, dev_delta_ptr);
    hipDeviceSynchronize();
    checkLastCudaError();

    gpu_time += wtime() - timing_gpu;

    //printf("Kernels complete for itter %d, updating data in CPU\n", loop);

    timing_transfers = wtime();
    /* TODO: Copy dev_delta_ptr to &delta
    checkCuda(hipMemcpy(...)); */
    checkCuda(hipMemcpy(&delta, dev_delta_ptr, sizeof(double), hipMemcpyDeviceToHost));
    transfers_time += wtime() - timing_transfers;

    const unsigned int update_centroids_block_sz = (numCoords * numClusters > blockSize) ? blockSize : numCoords *
                                                                                                       numClusters;  /* TODO: can use different blocksize here if deemed better */

    const unsigned int update_centroids_dim_sz = (numCoords * numClusters + update_centroids_block_sz - 1) / update_centroids_block_sz;; /*
         TODO: calculate dim for "update_centroids"*/
    timing_gpu = wtime();
    /* TODO: use dim for "update_centroids" and fire it
update_centroids<<< update_centroids_dim_sz, update_centroids_block_sz, 0 >>>
  (numCoords, numClusters, devicenewClusterSize, devicenewClusters, deviceClusters);  */
    update_centroids<<< update_centroids_dim_sz, update_centroids_block_sz, 0 >>>
    (numCoords, numClusters, devicenewClusterSize, devicenewClusters, deviceClusters); 
    hipDeviceSynchronize();
    checkLastCudaError();
    gpu_time += wtime() - timing_gpu;

    timing_cpu = wtime();
    delta /= numObjs;
    //printf("delta is %f - ", delta);
    loop++;
    //printf("completed loop %d\n", loop);
    cpu_time += wtime() - timing_cpu;

    timing_internal = wtime() - timing_internal;
    if (timing_internal < timer_min) timer_min = timing_internal;
    if (timing_internal > timer_max) timer_max = timing_internal;
  } while (delta > threshold && loop < loop_threshold);


  checkCuda(hipMemcpy(membership, deviceMembership,
                       numObjs * sizeof(int), hipMemcpyDeviceToHost));
  checkCuda(hipMemcpy(dimClusters[0], deviceClusters,
                       numClusters * numCoords * sizeof(double), hipMemcpyDeviceToHost));

  for (i = 0; i < numClusters; i++) {
    //if (newClusterSize[i] > 0) {
    for (j = 0; j < numCoords; j++) {
      clusters[i * numCoords + j] = dimClusters[j][i];
    }
    //}
  }

  timing = wtime() - timing;
  printf("nloops = %d  : total = %lf ms\n\t-> t_loop_avg = %lf ms\n\t-> t_loop_min = %lf ms\n\t-> t_loop_max = %lf ms\n\t"
         "-> t_cpu_avg = %lf ms\n\t-> t_gpu_avg = %lf ms\n\t-> t_transfers_avg = %lf ms\n\n|-------------------------------------------|\n",
         loop, 1000 * timing, 1000 * timing / loop, 1000 * timer_min, 1000 * timer_max,
         1000 * cpu_time / loop, 1000 * gpu_time / loop, 1000 * transfers_time / loop);

  char outfile_name[1024] = {0};
  sprintf(outfile_name, "Execution_logs/silver1-V100_Sz-%lu_Coo-%d_Cl-%d.csv",
          numObjs * numCoords * sizeof(double) / (1024 * 1024), numCoords, numClusters);
  FILE *fp = fopen(outfile_name, "a+");
  if (!fp) error("Filename %s did not open succesfully, no logging performed\n", outfile_name);
  fprintf(fp, "%s,%d,%lf,%lf,%lf\n", "All_GPU_Delta_Reduction", blockSize, timing / loop, timer_min, timer_max);
  fclose(fp);

  checkCuda(hipFree(deviceObjects));
  checkCuda(hipFree(deviceClusters));
  checkCuda(hipFree(devicenewClusters));
  checkCuda(hipFree(devicenewClusterSize));
  checkCuda(hipFree(deviceMembership));

  return;
}

